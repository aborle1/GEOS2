/*
* Copyright 1993-2006 NVIDIA Corporation.  All rights reserved.
*
* NOTICE TO USER:
*
* This source code is subject to NVIDIA ownership rights under U.S. and
* international Copyright laws.
*
* NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
* CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
* IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
* REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
* MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
* IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
* OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
* OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
* OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE
* OR PERFORMANCE OF THIS SOURCE CODE.
*
* U.S. Government End Users.  This source code is a "commercial item" as
* that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting  of
* "commercial computer software" and "commercial computer software
* documentation" as such terms are used in 48 C.F.R. 12.212 (SEPT 1995)
* and is provided to the U.S. Government only as a commercial end item.
* Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
* 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
* source code with only those rights set forth herein.
*/

/* This sample is a templatized version of the template project.
* It also shows how to correctly templatize dynamically allocated shared
* memory arrays.
* Host code.
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>

// includes, kernels
#include "simpleTemplates_kernel.cu"
#define BLOCKSIZE 256
template<class T> T*
nco_cuda_var_add /* [fnc] Add first operand to second operand */
( /* I [enm] netCDF type of operands */
 const long size, /* I [nbr] Size (in elements) of operands */
 const int has_mss_val, /* I [flg] Flag for missing values */
 T mss_val, /* I [flg] Value of missing value */
 T* op1, /* I [val] Values of first operand */
 T* op2) /* I/O [val] Values of second operand on input, values of sum on output */
{
  /* Purpose: Add value of first operand to value of second operand
     and store result in second operand. 
     Assume operands conform, are same type, and are in memory
     nco_var_add() does _not_ increment tally counter
     nco_var_add_tll_ncra() does increment tally counter */
  
  /* Addition is currently defined as op2:=op1+op2 */
 

  T* op_in_D, *op_out_D;GA
  hipMalloc((void**)&op_in_D, sizeof(T)*size);
  hipMalloc((void**)&op_out_D,sizeof(T)*size);
  
  hipMemcpy(op_in_D, op1, sizeof(T)*size,hipMemcpyHostToDevice);
  hipMemcpy(op_out_D, op2, sizeof(T)*size,hipMemcpyHostToDevice);

  uint gridsize=size%BLOCKSIZE ==0?size/BLOCKSIZE :size/BLOCKSIZE +1;
	uint blocksize=BLOCKSIZE;
	dim3 dimBlock(blocksize);
	dim3 dimGrid(gridsize);

  testKernel<T><<<gridsize,blocksize>>>(op_in_D, op_out_D, size, mss_val, has_mss_val);
  hipMemcpy(op2, op_out_D, sizeof(T)*size,hipMemcpyDeviceToHost);

  return op2;
    
} /* end nco_var_add() */
